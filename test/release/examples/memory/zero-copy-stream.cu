#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>

__global__
void saxpy(int n, float a, float* x, float* y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__
void saxpy1(int n, float a, float* x, float* y, float* z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i] + z[i];
}


int main(int argc, char *argv[]){

  //int N = 1 << std::stoi(argv[1]);
  int N = std::pow(2, std::stof(argv[1]));
  printf("N: %d\n", N);
  float *hx, *hy;
 // float *dx, *dy;

  //hx = (float*)malloc(N*sizeof(float));
  //hy = (float*)malloc(N*sizeof(float));

  //try different flag like hipHostMallocDefault
  auto err1 = hipHostAlloc((void**)&hx, N*sizeof(float), hipHostMallocMapped);
  auto err2 = hipHostAlloc((void**)&hy, N*sizeof(float), hipHostMallocMapped);
  //auto err3 = hipHostAlloc((void**)&hz, N*sizeof(float), hipHostMallocMapped);

  printf("%d\n", (int)err1);
  printf("%d\n", (int)err2);
  //printf("%d\n", (int)err3);
  //hipMalloc(&d_x, N*sizeof(float));
  //hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    hx[i] = 1.0f;
    hy[i] = 3.0f;
    //hz[i] = 2.0f;
  }

  //hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  //hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  //hipHostGetDevicePointer(&dx, hx, 0);
  //hipHostGetDevicePointer(&dy, hy, 0);
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, dx, dy);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, hx, hy);
  //saxpy1<<<(N+255)/256, 256>>>(N, 2.0f, hx, hy, hz);
  hipDeviceSynchronize();//needs to be called after kernel is finished
  //hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(hy[i]-7.0f));
  printf("Max error: %f\n", maxError);

  hipFree(hx);
  hipFree(hy);
  //hipFree(hz);
  //hipFree(d_x);
  //hipFree(d_y);
  //free(x);
  //free(y);

}
