
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <cstdlib>
#include <cmath>

__global__
void saxpy(int n, float a, float* x, float* y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__
void saxpy1(int n, float a, float* x, float* y, float* z, float* x1, float* y1)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i] + z[i] + x1[i] + y1[i];
}


int main(int argc, char *argv[]){

  int N = std::pow(2, std::stof(argv[1]));//1 << std::stoi(argv[1]);
  printf("N: %d\n", N);
  float *hx, *hy, *hz, *x1, *y1;
 // float *dx, *dy;

  //hx = (float*)malloc(N*sizeof(float));
  //hy = (float*)malloc(N*sizeof(float));

  //try different flag like cudaHostAllocDefault
  auto err1 = hipMallocManaged((void**)&hx, N*sizeof(float));
  auto err2 = hipMallocManaged((void**)&hy, N*sizeof(float));
  auto err3 = hipMallocManaged((void**)&hz, N*sizeof(float));
  auto err4 = hipMallocManaged((void**)&x1, N*sizeof(float));
  auto err5 = hipMallocManaged((void**)&y1, N*sizeof(float));

  printf("%d\n", (int)err1);
  printf("%d\n", (int)err2);
  printf("%d\n", (int)err3);
  printf("%d\n", (int)err4);
  printf("%d\n", (int)err5);
  //printf("%d\n", (int)err3);
  //cudaMalloc(&d_x, N*sizeof(float));
  //cudaMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    hx[i] = rand();
    hy[i] = rand();
    hz[i] = rand();
    x1[i] = rand();
    y1[i] = rand();
   // hz[i] = 2.0f;
  }

  //cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
  //cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  //cudaHostGetDevicePointer(&dx, hx, 0);
  //cudaHostGetDevicePointer(&dy, hy, 0);
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, dx, dy);
  saxpy1<<<(N+255)/256, 256>>>(N, 2.0f, hx, hy, hz, x1, y1);
  //saxpy1<<<(N+255)/256, 256>>>(N, 2.0f, hx, hy, hz);
  hipDeviceSynchronize();//needs to be called after kernel is finished
  //cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

  //float maxError = 0.0f;
  //for (int i = 0; i < N; i++)
  //  maxError = max(maxError, abs(hy[i]-7.0f));
  //printf("Max error: %f\n", maxError);

  hipFree(hx);
  hipFree(hy);
  hipFree(hz);
  hipFree(x1);
  hipFree(y1);
  //cudaFree(hz);
  //cudaFree(d_x);
  //cudaFree(d_y);
  //free(x);
  //free(y);

}
