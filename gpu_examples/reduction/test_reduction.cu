#include <hip/hip_runtime.h>
#include <iostream>

#define THREADS_PER_BLOCK 256


__global__
void reduce2(float *g_idata, float *g_odata) {

  extern __shared__ float sdata[];
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = g_idata[i];

  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }

    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(){

int N = 1<<20;
float *x, *y, *d_x, *d_y;

x = (float*)malloc(N*sizeof(float));
y = (float*)malloc(N*sizeof(float));

hipMalloc(&d_x, N*sizeof(float));
hipMalloc(&d_y, N*sizeof(float));

for (int i = 0; i < N; i++) {
  x[i] = 1.0f;
  //y[i] = 2.0f;
}

hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

//<<<numBlocks, numThreads, smemSize>>
//reduce2<<<1,32, N*sizeof(float)>>>(d_x, d_y);
reduce2<<<(N+255)/256, 256, 256*sizeof(float)>>>(d_x, d_y);
//add<<<1, 1>>>(N, d_x, d_y);

hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

float total_sum = 0.0f;

//reduction for each block
for(int i = 0; i < (N / 256); i++){
  total_sum += y[i];
}

std::cout << total_sum <<std::endl;

hipFree(d_x);
hipFree(d_y);
free(x);
free(y);

}
