#include "hip/hip_runtime.h"
__global__ void reduce2(int *g_idata, int *g_odata) {

  extern__shared__int sdata[];
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = g_idata[i];

  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }

    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
