
#include <hip/hip_runtime.h>
#include <stdio.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

//check errors
void postprocess(const float *ref, const float *res, int n)
{
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if(passed){
    printf("test passed");
  }
}

__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

int main(){

  const int nx = 1024;
  const int ny = 1024;
  const int mem_size = nx*ny*sizeof(float);

  dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  float *h_idata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold    = (float*)malloc(mem_size);

  float *d_idata, *d_tdata;
  hipMalloc(&d_idata, mem_size);
  hipMalloc(&d_tdata, mem_size);

  // host
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      h_idata[j*nx + i] = j*nx + i;


  // correct result for error checking
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      gold[j*nx + i] = h_idata[i*nx + j];

  hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

  hipMemset(d_tdata, 0, mem_size); 
  transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost);
  postprocess(gold, h_tdata, nx * ny);

  hipFree(d_tdata);
  hipFree(d_idata);
  free(h_idata);
  free(h_tdata);
  free(gold);


}

